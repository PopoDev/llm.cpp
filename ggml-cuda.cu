#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>

#include "ggml-cuda.h"
#include "ggml.h"


#define CUDA_ADD_BLOCK_SIZE 256
#define CUDA_MUL_BLOCK_SIZE 256
#define CUDA_SILU_BLOCK_SIZE 256
#define CUDA_CPY_BLOCK_SIZE 32
#define CUDA_SCALE_BLOCK_SIZE 256
#define CUDA_ROPE_BLOCK_SIZE 256
#define CUDA_DIAG_MASK_INF_BLOCK_SIZE 32

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

static void ggml_cuda_mul_mat_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy data to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}


static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}