#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <atomic>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

#if defined(_MSC_VER)
#pragma warning(disable: 4244 4267) // possible loss of data
#endif

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#if CUDART_VERSION >= 12
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d: %s\n",                         \
                    err_, __FILE__, __LINE__, cublasGetStatusString(err_));             \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#else
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);  \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#endif // CUDART_VERSION >= 11

typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, float & v0, float & v1);
typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);
typedef void (*to_fp16_cuda_t)(const void * x, half * y, int k, hipStream_t stream);
typedef void (*dot_kernel_k_t)(const void * vx, const int ib, const int iqs, const float * y, float & v);
typedef void (*cpy_kernel_t)(const char * cx, char * cdst);
typedef void (*ggml_cuda_func_t)(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst);
typedef void (*ggml_cuda_op_t)(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i, float * src0_ddf_i,
    float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main);

#define WARP_SIZE 32

#define CUDA_ADD_BLOCK_SIZE 256
#define CUDA_MUL_BLOCK_SIZE 256
#define CUDA_SILU_BLOCK_SIZE 256
#define CUDA_CPY_BLOCK_SIZE 32
#define CUDA_SCALE_BLOCK_SIZE 256
#define CUDA_ROPE_BLOCK_SIZE 256
#define CUDA_DIAG_MASK_INF_BLOCK_SIZE 32
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256

static __global__ void add_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] + y[i];
}

static __global__ void mul_f32(const float * x, const float * y, float * dst, const int kx, const int ky) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= kx) {
        return;
    }
    dst[i] = x[i] * y[i%ky];
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void rms_norm_f32(const float * x, float * dst, const int ncols) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const float eps = 1e-6;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row*ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    const float mean = tmp / ncols;
    const float scale = 1.0f / sqrtf(mean + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row*ncols + col] = scale * x[row*ncols + col];
    }
}

static __global__ void mul_mat_p021_f16_f32(const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int nchannels_x) {
    const half * x = (half *) vx;

    const int row_x = blockDim.y*blockIdx.y + threadIdx.y;
    const int channel = blockDim.z*blockIdx.z + threadIdx.z;

    const int nrows_y = ncols_x;
    const int nrows_dst = nrows_x;
    const int row_dst = row_x;

    float tmp = 0.0f;

    for (int col_x0 = 0; col_x0 < ncols_x; col_x0 += blockDim.x) {
        const int col_x = col_x0 + threadIdx.x;

        if (col_x >= ncols_x) {
            break;
        }

        // x is transposed and permuted
        const int ix = row_x*nchannels_x*ncols_x + channel*ncols_x + col_x;
        const float xi = __half2float(x[ix]);

        const int row_y = col_x;


        // y is not transposed but permuted
        const int iy = channel*nrows_y + row_y;

        tmp += xi * y[iy];
    }

    // dst is not transposed and not permuted
    const int idst = channel*nrows_dst + row_dst;

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[idst] = tmp;
    }
}

static __global__ void mul_mat_vec_nc_f16_f32( // nc == non-contiguous
    const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x,
    const int row_stride_x, const int nchannels_x, const int channel_stride_x) {

    const half * x = (half *) vx;

    const int row_x = blockDim.y*blockIdx.y + threadIdx.y;
    const int channel = blockDim.z*blockIdx.z + threadIdx.z;

    const int nrows_y = ncols_x;
    const int nrows_dst = nrows_x;
    const int row_dst = row_x;

    const int idst = channel*nrows_dst + row_dst;

    float tmp = 0.0f;

    for (int col_x0 = 0; col_x0 < ncols_x; col_x0 += blockDim.x) {
        const int col_x = col_x0 + threadIdx.x;

        if (col_x >= ncols_x) {
            break;
        }

        const int ix = channel*channel_stride_x + row_x*row_stride_x + col_x;
        const float xi = __half2float(x[ix]);

        const int row_y = col_x;

        const int iy = channel*nrows_y + row_y;

        tmp += xi * y[iy];
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[idst] = tmp;
    }
}

static __device__ void cpy_1_f32_f32(const char * cxi, char * cdsti) {
    const float * xi = (float *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f32_f16(const char * cxi, char * cdsti) {
    const float * xi = (float *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = __float2half(*xi);
}

template <cpy_kernel_t cpy_1>
static __global__ void cpy_f32_f16(const char * cx, char * cdst, const int ne,
                                   const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
                                   const int ne10, const int ne11, const int nb10, const int nb11, const int nb12) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    // determine indices i02/i12, i01/i11, i00/i10 as a function of index i of flattened tensor
    // then combine those indices with the corresponding byte offsets to get the total offsets
    const int i02 = i / (ne00*ne01);
    const int i01 = (i - i02*ne01*ne00) / ne00;
    const int i00 = i - i02*ne01*ne00 - i01*ne00;
    const int x_offset = i00*nb00 + i01*nb01 + i02*nb02;

    const int i12 = i / (ne10*ne11);
    const int i11 = (i - i12*ne10*ne11) / ne10;
    const int i10 = i - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = i10*nb10 + i11*nb11 + i12*nb12;

    cpy_1(cx + x_offset, cdst + dst_offset);
}

// rope == RoPE == rotary positional embedding
static __global__ void rope_f32(const float * x, float * dst, const int ncols, const float p, const float theta_scale) {
    const int col = 2*(blockDim.x*blockIdx.x + threadIdx.x);

    if (col >= ncols) {
        return;
    }

    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int i = row*ncols + col;

    const float theta = p*powf(theta_scale, col/2);
    const float sin_theta = sinf(theta);
    const float cos_theta = cosf(theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + 1];

    dst[i + 0] = x0*cos_theta - x1*sin_theta;
    dst[i + 1] = x0*sin_theta + x1*cos_theta;
}

static __global__ void diag_mask_inf_f32(const float * x, float * dst, const int ncols, const int rows_per_channel, const int n_past) {
    const int col = blockDim.x*blockIdx.x + threadIdx.x;
    const int row = blockDim.y*blockIdx.y + threadIdx.y;

    if (col >= ncols) {
        return;
    }

    const int i = row*ncols + col;
    // dst[i] = col > n_past + row ? -INFINITY : x[i];
    dst[i] = x[i] - (col > n_past + row % rows_per_channel) * INT_MAX; // equivalent within rounding error but slightly faster on GPU
}

// the CUDA soft max implementation differs from the CPU implementation
// instead of doubles floats are used
// values are also not normalized to the maximum value by subtracting it in the exponential function
// theoretically these changes could cause problems with rounding error and arithmetic overflow but for LLaMa it seems to be fine
static __global__ void soft_max_f32(const float * x, float * dst, const int ncols) {
    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;

    float tmp = 0.0;

    for (int block_start = 0; block_start < ncols; block_start += block_size) {
        const int col = block_start + tid;

        if (col >= ncols) {
            break;
        }

        const int i = row*ncols + col;
        const float val = expf(x[i]);
        tmp += val;
        dst[i] = val;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    for (int block_start = 0; block_start < ncols; block_start += block_size) {
        const int col = block_start + tid;

        if (col >= ncols) {
            break;
        }

        const int i = row*ncols + col;
        dst[i] /= tmp;
    }
}

static __global__ void scale_f32(const float * x, float * dst, const float scale, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = scale * x[i];
}

static void add_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_ADD_BLOCK_SIZE - 1) / CUDA_ADD_BLOCK_SIZE;
    add_f32<<<num_blocks, CUDA_ADD_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void mul_f32_cuda(const float * x, const float * y, float * dst, const int kx, const int ky, hipStream_t stream) {
    const int num_blocks = (kx + CUDA_MUL_BLOCK_SIZE - 1) / CUDA_MUL_BLOCK_SIZE;
    mul_f32<<<num_blocks, CUDA_MUL_BLOCK_SIZE, 0, stream>>>(x, y, dst, kx, ky);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void rms_norm_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    rms_norm_f32<<<nrows, block_dims, 0, stream>>>(x, dst, ncols);
}

static void ggml_mul_mat_p021_f16_f32_cuda(const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int nchannels_x, hipStream_t stream) {
    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    mul_mat_p021_f16_f32<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols_x, nrows_x, nchannels_x);
}

static void ggml_mul_mat_vec_nc_f16_f32_cuda(
    const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int row_stride_x,
    const int nchannels_x, const int channel_stride_x, hipStream_t stream) {

    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    mul_mat_vec_nc_f16_f32<<<block_nums, block_dims, 0, stream>>>
        (vx, y, dst, ncols_x, nrows_x, row_stride_x, nchannels_x, channel_stride_x);
}

static void ggml_cpy_f32_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
    const int ne10, const int ne11, const int nb10, const int nb11, const int nb12, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, nb00, nb01, nb02, ne10, ne11, nb10, nb11, nb12);
}

static void ggml_cpy_f32_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
    const int ne10, const int ne11, const int nb10, const int nb11, const int nb12, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, nb00, nb01, nb02, ne10, ne11, nb10, nb11, nb12);
}

static void scale_f32_cuda(const float * x, float * dst, const float scale, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    scale_f32<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

static void rope_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, const float p, const float theta_scale, hipStream_t stream) {
    GGML_ASSERT(nrows % 2 == 0);
    const dim3 block_dims(2*CUDA_ROPE_BLOCK_SIZE, 1, 1);
    const int num_blocks_x = (ncols + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(num_blocks_x, nrows, 1);
    rope_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols, p, theta_scale);
}

static void diag_mask_inf_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
    const dim3 block_dims(CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, nrows_x, 1);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

static void soft_max_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, hipStream_t stream) {
    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums(1, nrows_x, 1);
    soft_max_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x);
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 256

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
    int access_count = 0;
};

static cuda_buffer g_cuda_buffer_pool[GGML_CUDA_MAX_DEVICES][MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));
    size_t min_size_diff = SIZE_MAX;
    size_t min_size_diff_ok = size * 0.05; // wiggle room
    cuda_buffer* best_fit = nullptr; // candidate pointer
    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.size >= size && b.ptr != nullptr) {
            size_t size_diff = b.size - size;
            if (size_diff < min_size_diff) {
                best_fit = &b;
                min_size_diff = size_diff;
                if (size_diff < min_size_diff_ok) {
                    break;
                }
            }
        }
    }
    if (best_fit != nullptr) {
        *actual_size = best_fit->size;
        void * ptr = best_fit->ptr;
        best_fit->ptr = nullptr;
        // best_fit->size = 0;
        best_fit->access_count++;
        return ptr;
    }
    //printf("CUDA MALLOC: Allocated MB: %.2f\n", (float)size/1024/1024);
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;    
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size; // the original size should still be correct
            b.access_count = 1;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

// unallocates any "free" buffers that have not been used (or less than n times since last free)
// for example call after evaluation
int ggml_cuda_pool_purge_buffers_with_access_count(int min_access_count, int device_id) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    int total_purged = 0;

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[device_id][i];
        if (b.ptr != nullptr && b.access_count < min_access_count) {
            if (id != device_id) {
                CUDA_CHECK(hipSetDevice(device_id));
            }
            CUDA_CHECK(hipFree(b.ptr));
            //printf("\n-----> CUDA: access count - purged buffer %d of size %zu for device %d\n", i, b.size, device_id);
            b.ptr = nullptr;
            b.size = 0;
            b.access_count = 0;
            
            total_purged++;
        }
    }
    return total_purged;
}
// resets access_count for all free buffers (for example before evaluation)
void ggml_cuda_pool_reset_all_counters(int device_id) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[device_id][i];
        if (b.ptr != nullptr) {
            b.access_count = 0;
            //printf("CUDA: reset buffer %d of size %zu access_count %d for device %d\n", i, b.size, b.access_count, device_id);
        }
    }
}


static void * g_scratch_buffer = nullptr;
static size_t g_scratch_size = 1024*1024*1024; // 1 GB by default
static size_t g_scratch_offset = 0;

#define GGML_CUDA_MAX_STREAMS 8 // Set this to 1 for reproducible matrix multiplication.
#define GGML_CUDA_MAX_EVENTS 64

// Note: tensor_split defines the breakpoints of tensors that can be split {0,0.5}
static float g_tensor_split[GGML_CUDA_MAX_DEVICES] = {0};
static GPUStatus g_system_gpu_status;

static hipblasHandle_t g_cublas_handles[GGML_CUDA_MAX_DEVICES] = {nullptr};

static hipStream_t g_cudaStreams_main[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_STREAMS] = { nullptr };

static hipStream_t g_cudaStreams_memcpy_src1[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents_memcpy_src1[GGML_CUDA_MAX_DEVICES][GGML_CUDA_MAX_EVENTS] = { nullptr };

// Todo verify: free and total memory reported by hipMemGetInfo differs from gpu_z which also differs from hwinfo64.
// Update the system status about available GPUs and memory usage
void ggml_cuda_update_gpu_status(int device_id) {
    int currentDevice = 0;
    CUDA_CHECK(hipGetDevice(&currentDevice));
    if (device_id == -1) {
        // Update all devices 
        if (g_system_gpu_status.num_devices == 0)
        {
            CUDA_CHECK(hipGetDeviceCount(&g_system_gpu_status.num_devices));
            if (g_system_gpu_status.num_devices > GGML_CUDA_MAX_DEVICES) {
                g_system_gpu_status.num_devices = GGML_CUDA_MAX_DEVICES;
                fprintf(stderr, "WARNING: GGML_CUDA_MAX_DEVICES is smaller than the number of devices on the system. Using first %d devices.\n", GGML_CUDA_MAX_DEVICES);
            }
            if (g_system_gpu_status.num_devices > g_system_gpu_status.max_gpus)
                    g_system_gpu_status.num_devices = g_system_gpu_status.max_gpus;
                
            g_system_gpu_status.total_vram = 0;
            for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
                CUDA_CHECK(hipGetDeviceProperties(&g_system_gpu_status.device_props[id], id));
            }
        }
        g_system_gpu_status.total_vram = 0;
        g_system_gpu_status.total_free_vram = 0;
        for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
            CUDA_CHECK(hipSetDevice(id));
            CUDA_CHECK(hipMemGetInfo(&g_system_gpu_status.device_vram_free[id], &g_system_gpu_status.device_vram_total[id]));
            g_system_gpu_status.total_vram += g_system_gpu_status.device_vram_total[id];
            g_system_gpu_status.total_free_vram += g_system_gpu_status.device_vram_free[id];
        }
        // restore current device
        if (currentDevice != g_system_gpu_status.num_devices-1) {
            CUDA_CHECK(hipSetDevice(currentDevice));
        }
    } else {
        // Update only the specified device
        CUDA_CHECK(hipGetDeviceProperties(&g_system_gpu_status.device_props[device_id], device_id));
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipMemGetInfo(&g_system_gpu_status.device_vram_free[device_id], &g_system_gpu_status.device_vram_total[device_id]));
        // go through all devices and update total/free
        g_system_gpu_status.total_vram = 0;
        g_system_gpu_status.total_free_vram = 0;
        for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
            g_system_gpu_status.total_vram += g_system_gpu_status.device_vram_total[id];
            g_system_gpu_status.total_free_vram += g_system_gpu_status.device_vram_free[id];
        }
        // restore current device
        if (device_id != currentDevice) {
            CUDA_CHECK(hipSetDevice(currentDevice));
        }
    }
    
#if 1
    // required for proper vram distribution but split tensors require memory on primary GPU which could be disabled
    // remove unused GPUs from available calculation
    bool all_zero = true;
    for (int i = 0; i < g_system_gpu_status.num_devices; ++i) {
        if (g_tensor_split[i] != 0.0f) {
            all_zero = false;
        }
    }
    if (!all_zero)
    for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
        if (g_tensor_split[id] >= 1.0 || (id > 0 && g_tensor_split[id] == g_tensor_split[id-1])) {
            g_system_gpu_status.total_vram -= g_system_gpu_status.device_vram_total[id];
            g_system_gpu_status.total_free_vram -= g_system_gpu_status.device_vram_free[id];
        }
        
    }
#endif
}
void ggml_cuda_print_gpu_status(const GPUStatus *status, bool print_summary) {
    if (status == NULL) {
        fprintf(stderr,"Error: Invalid GPU status pointer.\n");
        return;
    }

    const char *divider = "+----+------------------------------------+------------+-----------+-----------+-----------+-----------+";
    fprintf(stderr,"%s\n", divider);
    fprintf(stderr,"| ID | %-25s %2d found | %10s | %9s | %9s | %9s | %9s |\n", "Device", status->num_devices, "VRAM Total", "VRAM Free", "VRAM Used","Split at ", "Device");
    fprintf(stderr,"%s\n", divider);

    for (int i = 0; i < status->num_devices; ++i) {
        const struct hipDeviceProp_t *prop = &status->device_props[i];
        size_t vram_used = status->device_vram_total[i] - status->device_vram_free[i];
        float split_at_percentage = g_tensor_split[i] * 100;
        fprintf(stderr,"| %2d | %-34s | %7zu MB | %6zu MB | %6zu MB | %8.1f%% | %9s |\n", 
                i,prop->name, status->device_vram_total[i] / (1024 * 1024), status->device_vram_free[i] / (1024 * 1024), vram_used / (1024 * 1024),split_at_percentage, (i == status->main_device_id) ? "Primary" : "Secondary");
        // printf("%s\n", divider);
    }
    if (print_summary && status->num_devices > 1)
    {
        fprintf(stderr,"%s\n", divider);
        fprintf(stderr,"|    | %-34s | %7zu MB | %6zu MB | %6zu MB | %9s | %9s |\n", 
            "Device summary", status->total_vram / (1024 * 1024), status->total_free_vram / (1024 * 1024), (status->total_vram - status->total_free_vram) / (1024 * 1024), "N/A", "All");
    }
    fprintf(stderr,"%s\n", divider);
    
}

const GPUStatus* ggml_cuda_get_system_gpu_status(void) {
    return &g_system_gpu_status;
}
void ggml_cuda_set_max_gpus(int max_gpus) {
    g_system_gpu_status.max_gpus = max_gpus;
}

// can be called multithreaded to prevent 1-2 seconds delay on handle creation
bool ggml_init_cublas(bool check_only) {
    static volatile bool initialized = false;
    if (check_only || initialized) return initialized;
        
    int currentDevice = 0;
    CUDA_CHECK(hipGetDevice(&currentDevice));
    if (!initialized) {
        //g_system_gpu_status.num_devices = 0;
        if (g_system_gpu_status.num_devices == 0)
            ggml_cuda_update_gpu_status(-1);

        bool all_zero = true;
        for (int i = 0; i < g_system_gpu_status.num_devices; ++i) {
            if (g_tensor_split[i] != 0.0f) {
                all_zero = false;
            }
        }
        if (all_zero)
        {
            int64_t total_vram = 0;
            for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
                g_tensor_split[id] = total_vram;
                size_t vram_free;
                // vram_total = g_system_gpu_status.device_vram_total[id];
                vram_free = g_system_gpu_status.device_vram_free[id];
                total_vram += vram_free;
            }
            for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
                g_tensor_split[id] /= total_vram;
            }
        }
        //ggml_cuda_print_gpu_status(&g_system_gpu_status,true);
        // printf("Preparing CUDA for %d devices: ",g_system_gpu_status.num_devices);
        for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
            CUDA_CHECK(hipSetDevice(id));

            // create streams
            for (int i = 0; i < GGML_CUDA_MAX_STREAMS; ++i) {
                CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_main[id][i], hipStreamNonBlocking));
                CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_memcpy_src1[id][i], hipStreamNonBlocking));
            }
            // create events
            for (int i = 0; i < GGML_CUDA_MAX_EVENTS; ++i) {
                CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents_memcpy_src1[id][i], hipEventDisableTiming));
            }

            // create cublas handle
            CUBLAS_CHECK(hipblasCreate(&g_cublas_handles[id]));
            CUBLAS_CHECK(hipblasSetMathMode(g_cublas_handles[id], HIPBLAS_TF32_TENSOR_OP_MATH));
        }
        CUDA_CHECK(hipSetDevice(currentDevice));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
        
    }
    return initialized;
}

// prepare tensor split before we've initizalized cublas
void ggml_cuda_set_tensor_split_prepare(const float * tensor_split, int num_devices) {
    int cur_num_devices = g_system_gpu_status.num_devices;
    g_system_gpu_status.num_devices = num_devices;
    ggml_cuda_set_tensor_split(tensor_split);
    g_system_gpu_status.num_devices = cur_num_devices;
}
// expect array of float proportions where to split each device id, if all 0.0 then no change to default split
void ggml_cuda_set_tensor_split(const float * tensor_split) {
    bool all_zero = true;
    for (int i = 0; i < g_system_gpu_status.num_devices; ++i) {
        if (tensor_split[i] != 0.0f) {
            all_zero = false;
            break;
        }
    }
    if (all_zero) {
        return;
    }
    float split_sum = 0.0f;
    for (int i = 0; i < g_system_gpu_status.num_devices; ++i) {
        g_tensor_split[i] = split_sum;
        split_sum += tensor_split[i];
    }
    for (int i = 0; i < g_system_gpu_status.num_devices; ++i) {
        float device_prop = tensor_split[i] / split_sum;
        if (device_prop == 0.0f) {
            g_tensor_split[i] = 1.0f;
        }
        else {
            g_tensor_split[i] /= split_sum;
        }

    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }
    if (g_system_gpu_status.num_devices == 0) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        // The allocation error can be bypassed. A null ptr will assigned out of this function.
        // This can fixed the OOM error in WSL.
        hipGetLastError();
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned (CUDA optimized) memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_cpy_tensor_2d(
    void * dst, const struct ggml_tensor * src, int64_t i3, int64_t i2, int64_t i1_low, int64_t i1_high, hipStream_t stream) {

    hipMemcpyKind kind;
    char * src_ptr;
    if (src->backend == GGML_BACKEND_CPU) {
        kind = hipMemcpyHostToDevice;
        src_ptr = (char *) src->data;
    } else if (src->backend == GGML_BACKEND_GPU) {
        kind = hipMemcpyDeviceToDevice;
        struct ggml_tensor_extra_gpu * extra = (ggml_tensor_extra_gpu *) src->extra;
        int id;
        CUDA_CHECK(hipGetDevice(&id));
        src_ptr = (char *) extra->data_device[id];
    } else {
        GGML_ASSERT(false);
    }
    char * dst_ptr = (char *) dst;

    const int64_t ne0 = src->ne[0];
    const int64_t nb0 = src->nb[0];
    const int64_t nb1 = src->nb[1];
    const int64_t nb2 = src->nb[2];
    const int64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const int64_t ts = ggml_type_size(type);
    const int64_t bs = ggml_blck_size(type);
    int64_t i1_diff = i1_high - i1_low;

    const char * x = src_ptr + i1_low*nb1 + i2*nb2 + i3*nb3;
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst_ptr, x, i1_diff*nb1, kind, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst_ptr, ts*ne0/bs, x, nb1, ts*ne0/bs, i1_diff, kind, stream);
    } else {
        for (int64_t i1 = 0; i1 < i1_diff; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) (dst_ptr + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, kind, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

inline void ggml_cuda_op_add(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne0 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    add_f32_cuda(src0_ddf_i, src1_ddf_i, dst_ddf_i, ne0*i01_diff, cudaStream_main);

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_mul(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    for (int64_t i01 = i01_low; i01 < i01_high; i01++) {
        const int64_t i11 = i1*ne11 + i01%ne11; // broadcast src1 across src0

        float * src0_ddf_i01 = src0_ddf_i + i01*ne00;
        float * src1_ddf_i01 = src1_ddf_i + i11*ne10;
        float * dst_ddf_i01 = dst_ddf_i + i01*ne00;

        // compute
        mul_f32_cuda(src0_ddf_i01, src1_ddf_i01, dst_ddf_i01, ne00, ne10, cudaStream_main);
    }

    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
}

inline void ggml_cuda_op_silu(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    silu_f32_cuda(src0_ddf_i, dst_ddf_i, ne00*i01_diff, cudaStream_main);

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_rms_norm(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    rms_norm_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, cudaStream_main);

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_mul_mat_cublas(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int64_t ne0 = dst->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    dst->meta.cuda_perf_mal_mul_type=32;
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    // the main device has a larger memory buffer to hold the results from all GPUs
    // ldc == nrows of the matrix that cuBLAS writes into
    int ldc = dst->backend == GGML_BACKEND_GPU && id == g_system_gpu_status.main_device_id ? ne0 : i01_diff;

    CUBLAS_CHECK(hipblasSetStream(g_cublas_handles[id], cudaStream_main));
    CUBLAS_CHECK(
        hipblasSgemm(g_cublas_handles[id], HIPBLAS_OP_T, HIPBLAS_OP_N,
                i01_diff, ne11, ne10,
                &alpha, src0_ddf_i, ne00,
                        src1_ddf_i, ne10,
                &beta,  dst_ddf_i,  ldc));

    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
    (void) i1;
}
__global__ void float_to_half(const float* src, __half* dst, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        dst[idx] = __float2half(src[idx]);
    }
}
// takes 16,32 bit returns 32 bit - internally converts src1 to 16 bit
inline void ggml_cuda_op_mul_mat_cublas_f16_f32(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    __half * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int64_t ne0 = dst->ne[0];
    const int64_t i01_diff = i01_high - i01_low;
    dst->meta.cuda_perf_mal_mul_type=16;
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    // we need to convert src1_ddf_i to half precision
    __half* src1_ddf_i_half;
    size_t src1_size = ne10 * ne11 * sizeof(__half);
    size_t actual_size = 0;
    src1_ddf_i_half = (half *)ggml_cuda_pool_malloc(src1_size,&actual_size);
    float_to_half<<<(ne10 * ne11 + 255) / 256, 256, 0, cudaStream_main>>>(src1_ddf_i, src1_ddf_i_half, ne10 * ne11);
    CUDA_CHECK(hipStreamSynchronize(cudaStream_main));

    // the main device has a larger memory buffer to hold the results from all GPUs
    // ldc == nrows of the matrix that cuBLAS writes into
    int ldc = dst->backend == GGML_BACKEND_GPU && id == g_system_gpu_status.main_device_id ? ne0 : i01_diff;

    CUBLAS_CHECK(hipblasSetStream(g_cublas_handles[id], cudaStream_main));
    CUBLAS_CHECK(
        hipblasGemmEx(g_cublas_handles[id], HIPBLAS_OP_T, HIPBLAS_OP_N,
                i01_diff, ne11, ne10,
                &alpha, src0_ddf_i, HIP_R_16F, ne00,
                        src1_ddf_i_half, HIP_R_16F, ne10,
                &beta,  dst_ddf_i,  HIP_R_32F, ldc,
                HIPBLAS_COMPUTE_32F_FAST_16F,
                HIPBLAS_GEMM_DEFAULT));

    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
    (void) i1;
    ggml_cuda_pool_free(src1_ddf_i_half,actual_size);
}
// src0 is actually fp16, everything else is same as normal
inline void ggml_cuda_op_mul_mat_cublas_f16_f32_wrapper(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i, float * src0_ddf_i,
    float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main) {

    // Cast float pointers to half-precision pointers
    __half * src0_ddf_i_half = reinterpret_cast<__half *>(src0_ddf_i);
    // __half * src1_ddf_i_half = reinterpret_cast<__half *>(src1_ddf_i);
    // __half * dst_ddf_i_half = reinterpret_cast<__half *>(dst_ddf_i);

    // Call the modified function with the casted pointers
    ggml_cuda_op_mul_mat_cublas_f16_f32(src0, src1, dst, src0_ddq_i, src0_ddf_i_half, src1_ddf_i, dst_ddf_i, i02, i01_low, i01_high, i1, cudaStream_main);
}

inline void ggml_cuda_op_rope(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) src1->data)[0];
    const int n_dims = ((int32_t *) src1->data)[1];
    const int mode   = ((int32_t *) src1->data)[2];
    GGML_ASSERT(mode == 0);

    const float theta_scale = powf((float)(dst->meta.i_custom[GGML_CUSTOM_I_ROPE_ANG_FREQ]?dst->meta.i_custom[GGML_CUSTOM_I_ROPE_ANG_FREQ]:10000), -2.0f/n_dims);
    float p = ((mode & 1) == 0 ? n_past + i02 : i02);
    // custom 2d rotation angle scale (needs a test - blind adapted)
    if (dst->meta.f_custom[GGML_CUSTOM_F_ROPE_ANG_SCALE] != 0.0f) {
        p *= dst->meta.f_custom[GGML_CUSTOM_F_ROPE_ANG_SCALE];
    }
    // compute
    rope_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, p, theta_scale, cudaStream_main);

    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_diag_mask_inf(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) src1->data)[0];

    // compute
    diag_mask_inf_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, ne01, n_past, cudaStream_main);

    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_soft_max(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    soft_max_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, cudaStream_main);

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_scale(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float scale = ((float *) src1->data)[0];

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    scale_f32_cuda(src0_ddf_i, dst_ddf_i, scale, ne00*i01_diff, cudaStream_main);

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

static void ggml_cuda_op(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                         ggml_cuda_op_t op, bool src0_needs_f32, bool flatten_rows) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];
    const int64_t nrows0 = ggml_nrows(src0);

    const bool use_src1 = src1 != nullptr;
    const int64_t ne10 = use_src1 ? src1->ne[0] : 1;
    const int64_t ne11 = use_src1 ? src1->ne[1] : 1;
    const int64_t ne12 = use_src1 ? src1->ne[2] : 1;
    const int64_t ne13 = use_src1 ? src1->ne[3] : 1;

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    GGML_ASSERT(dst->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(!use_src1 || src1->backend != GGML_BACKEND_GPU_SPLIT);

    // strides for iteration over dims 3 and 2
    const int64_t num_iters = flatten_rows ? 1 : ne02 * ne03;
    const int64_t stride_mod = flatten_rows ? ne02 * ne03 : 1;
    const int64_t src0_stride = ne00 * ne01 * stride_mod;
    const int64_t src1_stride = ne10 * ne11 * stride_mod;
    const int64_t dst_stride = ne0 * ne1 * stride_mod;

    const size_t src0_ts = ggml_type_size(src0->type);
    const size_t src0_bs = ggml_blck_size(src0->type);

    struct ggml_tensor_extra_gpu * src0_extra =            (ggml_tensor_extra_gpu *) src0->extra;
    struct ggml_tensor_extra_gpu * src1_extra = use_src1 ? (ggml_tensor_extra_gpu *) src1->extra : nullptr;
    struct ggml_tensor_extra_gpu * dst_extra  =            (ggml_tensor_extra_gpu *) dst->extra;

    const bool src0_on_device = src0->backend == GGML_BACKEND_GPU || src0->backend == GGML_BACKEND_GPU_SPLIT;
    const bool src0_is_contiguous = ggml_is_contiguous(src0);
    const bool src0_is_f32 = src0->type == GGML_TYPE_F32;

    const bool src1_is_contiguous = use_src1 && ggml_is_contiguous(src1);
    const bool src1_stays_on_host = use_src1 && (
        dst->op == GGML_OP_SCALE || dst->op == GGML_OP_DIAG_MASK_INF || dst->op == GGML_OP_ROPE);

    const bool split = src0->backend == GGML_BACKEND_GPU_SPLIT;

    // dd = data device
    char  * src0_ddq[GGML_CUDA_MAX_DEVICES] = {nullptr}; // quantized
    float * src0_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr}; // float
    float * src1_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};
    float *  dst_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};

    // asq = actual size quantized, asf = actual size float
    size_t src0_asq[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src0_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src1_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t  dst_asf[GGML_CUDA_MAX_DEVICES] = {0};

    for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
        if (!split && id != g_system_gpu_status.main_device_id) {
            continue;
        }

        const bool src1_on_device = use_src1 && src1->backend == GGML_BACKEND_GPU && id == g_system_gpu_status.main_device_id;
        const bool dst_on_device = dst->backend == GGML_BACKEND_GPU && id == g_system_gpu_status.main_device_id;

        int64_t row_low, row_high;
        if (split) {
            row_low = id == 0 ? 0 : nrows0*g_tensor_split[id];
            row_high = id == g_system_gpu_status.num_devices - 1 ? nrows0 : nrows0*g_tensor_split[id + 1];
        } else {
            row_low = 0;
            row_high = nrows0;
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t row_diff = row_high - row_low;

        hipSetDevice(id);

        if (src0_on_device && src0_is_contiguous) {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) src0_extra->data_device[id];
            } else {
                src0_ddq[id] = (char *) src0_extra->data_device[id];
            }
        } else {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * sizeof(float), &src0_asf[id]);
            } else {
                src0_ddq[id] = (char *) ggml_cuda_pool_malloc(row_diff*ne00 * src0_ts/src0_bs, &src0_asq[id]);
            }
        }

        if (use_src1 && !src1_stays_on_host) {
            if (src1_on_device && src1_is_contiguous) {
                src1_ddf[id] = (float *) src1_extra->data_device[id];
            } else {
                src1_ddf[id] = (float *) ggml_cuda_pool_malloc(num_iters*src1_stride * sizeof(float), &src1_asf[id]);
            }
        }
        if (dst_on_device) {
            dst_ddf[id] = (float *) dst_extra->data_device[id];
        } else {
            size_t size_dst_ddf = split ? row_diff*ne1 * sizeof(float) : num_iters*dst_stride * sizeof(float);
            dst_ddf[id] = (float *) ggml_cuda_pool_malloc(size_dst_ddf, &dst_asf[id]);
        }

        const int64_t i03_max = flatten_rows ? 1 : ne03;
        const int64_t i02_max = flatten_rows ? 1 : ne02;
        const int64_t rows_per_iter = flatten_rows ? nrows0 : ne01;

        for (int64_t i03 = 0; i03 < i03_max; i03++) {
            const int64_t i13 = i03 % ne13;
            for (int64_t i02 = 0; i02 < i02_max; i02++) {
                const int64_t i12 = i02 % ne12;

                const int64_t i0 = i03*ne02 + i02;

                // i0 values that contain the lower/upper rows for a split tensor when using multiple GPUs
                const int64_t i0_offset_low = row_low/rows_per_iter;
                const int64_t i0_offset_high = row_high/rows_per_iter;

                int64_t i01_low = 0;
                int64_t i01_high = rows_per_iter;
                if (split) {
                    if (i0 < i0_offset_low || i0 > i0_offset_high) {
                        continue;
                    }
                    if (i0 == i0_offset_low) {
                        i01_low = row_low % rows_per_iter;
                    }
                    if (i0 == i0_offset_high) {
                        i01_high = row_high % rows_per_iter;
                    }
                }

                // There is possibly a bug in the Windows nvcc compiler regarding instruction reordering or optimizing out local variables.
                // Removing the first assert or changing the order of the arguments causes the second assert to fail.
                // Removing both asserts results in i01_high becoming 0 which in turn results in garbage output.
                // The root cause seems to be a problem with i0_offset_high becoming 0 when it should always be >0 (for single GPU).
                GGML_ASSERT(i01_low == 0 || g_system_gpu_status.num_devices > 1);
                GGML_ASSERT(i01_high == rows_per_iter || g_system_gpu_status.num_devices > 1);

                const int64_t i01_diff = i01_high - i01_low;
                if (i01_diff == 0) {
                    continue;
                }
                const int64_t i11 = i13*ne12 + i12;

                hipStream_t cudaStream_main        =        g_cudaStreams_main[id][i0 % GGML_CUDA_MAX_STREAMS];
                hipStream_t cudaStream_memcpy_src1 = g_cudaStreams_memcpy_src1[id][i0 % GGML_CUDA_MAX_STREAMS];
                hipEvent_t  cudaEvent_memcpy_src1  =  g_cudaEvents_memcpy_src1[id][i0 % GGML_CUDA_MAX_EVENTS];

                // for split tensors the data begins at i0 == i0_offset_low
                char  * src0_ddq_i = src0_ddq[id] + (i0 - i0_offset_low)*src0_stride*src0_ts/src0_bs;
                float * src0_ddf_i = src0_ddf[id] + (i0 - i0_offset_low)*src0_stride;
                float * src1_ddf_i = src1_ddf[id] + i11*src1_stride;
                float * dst_ddf_i  =  dst_ddf[id] + (i0 - i0_offset_low)*dst_stride;

                // for split tensors the data pointer needs to be rounded down
                // to the bin edge for i03, i02 bins beyond the first
                if (i0 - i0_offset_low > 0) {
                    GGML_ASSERT(!flatten_rows);
                    src0_ddq_i -= (row_low % ne01)*ne00 * src0_ts/src0_bs;
                    src0_ddf_i -= (row_low % ne01)*ne00;
                    dst_ddf_i  -= (row_low % ne0)*ne1;
                }

                // the main device memory buffer can be on VRAM scratch, with space for all partial results
                // in that case an offset on dst_ddf_i is needed
                if (dst->backend == GGML_BACKEND_GPU && id == g_system_gpu_status.main_device_id) {
                    dst_ddf_i += i01_low; // offset is 0 if no tensor split
                }

                // copy src0, src1 to device if necessary
                if (use_src1 && !src1_stays_on_host) {
                    if (src1->backend == GGML_BACKEND_CPU) {
                        GGML_ASSERT(!flatten_rows || nrows0 == ggml_nrows(src1));
                        int64_t nrows1 = flatten_rows ? nrows0 : ne11;
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src1_ddf_i, src1, i03, i02, 0, nrows1, cudaStream_memcpy_src1));
                    } else if (src1->backend == GGML_BACKEND_GPU && src1_is_contiguous) {
                        if (id != g_system_gpu_status.main_device_id) {
                            GGML_ASSERT(!flatten_rows);
                            float * src1_ddf_i_source = (float *) src1_extra->data_device[g_system_gpu_status.main_device_id];
                            src1_ddf_i_source += i11*src1_stride;
                            CUDA_CHECK(hipMemcpyAsync(src1_ddf_i, src1_ddf_i_source, src1_stride*sizeof(float),
                                                    hipMemcpyDeviceToDevice, cudaStream_memcpy_src1));
                        }
                    } else if (src1_on_device && !src1_is_contiguous) {
                        GGML_ASSERT(!split);
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src1_ddf_i, src1, i03, i02, 0, ne11, cudaStream_main));
                    } else {
                        GGML_ASSERT(false);
                    }
                }
                CUDA_CHECK(hipEventRecord(cudaEvent_memcpy_src1, cudaStream_memcpy_src1));

                if (!src0_on_device || !src0_is_contiguous) {
                    if (src0_is_f32) {
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src0_ddf_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    } else {
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src0_ddq_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    }
                }

                // wait with main stream until src1 memcpy is done
                CUDA_CHECK(hipStreamWaitEvent(cudaStream_main, cudaEvent_memcpy_src1, 0));

                // do the computation
                op(src0, src1, dst, src0_ddq_i, src0_ddf_i, src1_ddf_i, dst_ddf_i, i02, i01_low, i01_high, i11, cudaStream_main);
                CUDA_CHECK(hipGetLastError());

                // copy dst to host or other device if necessary
                if (!dst_on_device) {
                    void * dst_off_device;
                    hipMemcpyKind kind;
                    if (dst->backend == GGML_BACKEND_CPU) {
                        dst_off_device = dst->data;
                        kind = hipMemcpyDeviceToHost;
                    } else if (dst->backend == GGML_BACKEND_GPU) {
                        dst_off_device = dst_extra->data_device[g_system_gpu_status.main_device_id];
                        kind = hipMemcpyDeviceToDevice;
                    } else {
                        GGML_ASSERT(false);
                    }
                    if (split) {
                        // src0 = weight matrix is saved as a transposed matrix for better memory layout.
                        // dst is NOT transposed.
                        // The outputs of cuBLAS matrix matrix multiplications can therefore NOT simply be concatenated for >1 GPU.
                        // Instead they need to be copied to the correct slice in ne0 = dst row index.
                        // If dst is a vector with ne0 == 1 then you don't have to do this but it still produces correct results.
                        for (int64_t j = 0; j < ne1; ++j) {
                            float * dhf_dst_i = (float *) ((char *) dst_off_device + (j*ne0 + i01_low)*sizeof(float) + i02*nb2 + i03*nb3);
                            CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i + j*i01_diff, i01_diff*sizeof(float), kind, cudaStream_main));
                        }
                    } else {
                        float * dhf_dst_i = (float *) ((char *) dst_off_device + i02*nb2 + i03*nb3);
                        CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i, dst_stride*sizeof(float), kind, cudaStream_main));
                    }
                }
            }
        }
    }

    // wait until each device is finished, then free their buffers
    for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
        if (src0_asq[id] == 0 && src0_asf[id] == 0 && src1_asf[id] == 0 && dst_asf[id] == 0) {
            continue;
        }

        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipDeviceSynchronize());

        if (src0_asq[id] > 0) {
            ggml_cuda_pool_free(src0_ddq[id], src0_asq[id]);
        }
        if (src0_asf[id] > 0) {
            ggml_cuda_pool_free(src0_ddf[id], src0_asf[id]);
        }
        if (src1_asf[id] > 0) {
            ggml_cuda_pool_free(src1_ddf[id], src1_asf[id]);
        }
        if (dst_asf[id] > 0) {
            ggml_cuda_pool_free(dst_ddf[id], dst_asf[id]);
        }
    }
}

void ggml_cuda_add(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_add, true, true);
}

void ggml_cuda_mul(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul, true, false); // TODO ggml_cuda_op needs modification for flatten
}

void ggml_cuda_silu(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_silu, true, true);
}

void ggml_cuda_rms_norm(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_rms_norm, true, true);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];
    // if cuda is disabled we reject
    if (g_system_gpu_status.num_devices == 0) {
        return false;
    }
    if (dst->meta.cuda_op_directive != -1) return dst->meta.cuda_op_directive? true : false;
    if (src0->meta.cuda_op_directive != -1) return src0->meta.cuda_op_directive? true : false;
    if (src1->meta.cuda_op_directive != -1) return src1->meta.cuda_op_directive? true : false;
    
    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        (ne0 >= 32 && ne1 >= 32 && ne10 >= 32)) {
            //todo: wouldn't it make sense to switch based on flops required  instead ?
            // printf("can_mul_mat: true for shape: %ld %ld %ld\n", ne0, ne1, ne10);
        return true;
    }

    return false;
}

void ggml_cuda_mul_mat_vec_p021(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst){
    GGML_ASSERT(ggml_is_permuted(src0) && ggml_is_permuted(src1));
    GGML_ASSERT(src0->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(src0->nb[0] <= src0->nb[1] && src0->nb[2] <= src0->nb[3]); // 0213 permutation
    GGML_ASSERT(src1->nb[0] <= src1->nb[1] && src1->nb[2] <= src1->nb[3]); // 0213 permutation
    GGML_ASSERT(src0->type == GGML_TYPE_F16);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    CUDA_CHECK(hipSetDevice(g_system_gpu_status.main_device_id));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_system_gpu_status.main_device_id][0];

    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    void * src0_ddq = src0_extra->data_device[g_system_gpu_status.main_device_id];

    struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;
    float * src1_ddf = (float *) src1_extra->data_device[g_system_gpu_status.main_device_id];

    struct ggml_tensor_extra_gpu * dst_extra = (ggml_tensor_extra_gpu *) dst->extra;
    float * dst_ddf = (float *) dst_extra->data_device[g_system_gpu_status.main_device_id];

    ggml_mul_mat_p021_f16_f32_cuda(src0_ddq, src1_ddf, dst_ddf, ne00, ne01, ne02, cudaStream_main);

    CUDA_CHECK(hipDeviceSynchronize());
}

void ggml_cuda_mul_mat_vec_nc(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst){
    GGML_ASSERT(!ggml_is_contiguous(src0) && ggml_is_contiguous(src1));
    GGML_ASSERT(!ggml_is_permuted(src0));
    GGML_ASSERT(src0->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(src0->type == GGML_TYPE_F16);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    CUDA_CHECK(hipSetDevice(g_system_gpu_status.main_device_id));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_system_gpu_status.main_device_id][0];

    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    void * src0_ddq = src0_extra->data_device[g_system_gpu_status.main_device_id];

    struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;
    float * src1_ddf = (float *) src1_extra->data_device[g_system_gpu_status.main_device_id];

    struct ggml_tensor_extra_gpu * dst_extra = (ggml_tensor_extra_gpu *) dst->extra;
    float * dst_ddf = (float *) dst_extra->data_device[g_system_gpu_status.main_device_id];

    const int row_stride_x = nb01 / sizeof(half);
    const int channel_stride_x = nb02 / sizeof(half);

    ggml_mul_mat_vec_nc_f16_f32_cuda(src0_ddq, src1_ddf, dst_ddf, ne00, ne01, row_stride_x, ne02, channel_stride_x, cudaStream_main);

    CUDA_CHECK(hipDeviceSynchronize());
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    bool all_on_device = (src0->backend == GGML_BACKEND_GPU || src0->backend == GGML_BACKEND_GPU_SPLIT) &&
        src1->backend == GGML_BACKEND_GPU && dst->backend == GGML_BACKEND_GPU;

    dst->meta.cuda_perf_mal_mul_type=1;
    if (all_on_device && ggml_is_permuted(src0) && ggml_is_permuted(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_p021(src0, src1, dst);
    } else if (all_on_device && !ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_nc(src0, src1, dst);
    }else if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul_mat_cublas, true, false);
    } else {
        GGML_ASSERT(false);
    }
}

void ggml_cuda_scale(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_scale, true, true);
}

void ggml_cuda_cpy(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(src0->backend == GGML_BACKEND_GPU);
    GGML_ASSERT(src1->backend == GGML_BACKEND_GPU);

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];

    CUDA_CHECK(hipSetDevice(g_system_gpu_status.main_device_id));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_system_gpu_status.main_device_id][0];

    const struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    const struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;

    char * src0_ddc = (char *) src0_extra->data_device[g_system_gpu_status.main_device_id];
    char * src1_ddc = (char *) src1_extra->data_device[g_system_gpu_status.main_device_id];

    if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f32_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f32_f16_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else {
        GGML_ASSERT(false);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    (void) dst;
}

void ggml_cuda_diag_mask_inf(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_diag_mask_inf, true, true);
}

void ggml_cuda_soft_max(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_soft_max, true, true);
}

void ggml_cuda_rope(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_rope, true, false); // FIXME flatten changes results
}

void ggml_cuda_nop(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    (void) src0;
    (void) src1;
    (void) dst;
}

// copy tensor (*data) to the correct device(s) and assign tensor->extra->data_device[] with cuda mem pointer
void ggml_cuda_transform_tensor(void * data, struct ggml_tensor * tensor) {
    int nrows = ggml_nrows(tensor);
    const size_t nb1 = tensor->nb[1];
    ggml_backend backend = tensor->backend;
    struct ggml_tensor_extra_gpu * extra = new struct ggml_tensor_extra_gpu;
    memset(extra, 0, sizeof(*extra));

    for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
        if (backend == GGML_BACKEND_GPU && id != g_system_gpu_status.main_device_id) {
            continue;
        }

        hipSetDevice(id);

        int row_low, row_high;
        if (backend == GGML_BACKEND_GPU) {
            row_low = 0;
            row_high = nrows;
        } else if (backend == GGML_BACKEND_GPU_SPLIT) {
            row_low = id == 0 ? 0 : nrows*g_tensor_split[id];
            row_high = id == g_system_gpu_status.num_devices - 1 ? nrows : nrows*g_tensor_split[id + 1];
        } else {
            GGML_ASSERT(false);
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t nrows_split = row_high - row_low;

        const size_t offset_split = row_low*nb1;
        const size_t size = ggml_nbytes_split(tensor, nrows_split);

        void * buf;
        CUDA_CHECK(hipMalloc(&buf, size));
        void * buf_host = (char*)data + offset_split;

        hipMemcpy(buf, buf_host, size, hipMemcpyHostToDevice);

        extra->data_device[id] = buf;
    }

    tensor->extra = extra;
}

void ggml_cuda_free_data(struct ggml_tensor * tensor) {
    if (tensor->backend != GGML_BACKEND_GPU && tensor->backend != GGML_BACKEND_GPU_SPLIT) {
        return;
    }

    ggml_tensor_extra_gpu * extra = (ggml_tensor_extra_gpu *) tensor->extra;

    for (int id = 0; id < g_system_gpu_status.num_devices; ++id) {
        if (extra->data_device[id] == nullptr) {
            continue;
        }

        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipFree(extra->data_device[id]));
    }

    delete extra;
}

void ggml_cuda_assign_buffers_impl(struct ggml_tensor * tensor, bool scratch) {
    if (scratch && g_scratch_size == 0) {
        return;
    }

    // recursively assign CUDA buffers until a compute tensor is found
    if (tensor->src0 != nullptr && tensor->src0->backend == GGML_BACKEND_CPU) {
        const ggml_op src0_op = tensor->src0->op;
        if (src0_op == GGML_OP_RESHAPE || src0_op == GGML_OP_TRANSPOSE || src0_op == GGML_OP_VIEW) {
            ggml_cuda_assign_buffers_impl(tensor->src0, scratch);
        }
    }
    if (tensor->op == GGML_OP_CPY && tensor->src1->backend == GGML_BACKEND_CPU) {
        ggml_cuda_assign_buffers_impl(tensor->src1, scratch);
    }

    tensor->backend = GGML_BACKEND_GPU;
    struct ggml_tensor_extra_gpu * extra = new ggml_tensor_extra_gpu;

    const bool inplace = (tensor->src0 != nullptr && tensor->src0->data == tensor->data) ||
        tensor->op == GGML_OP_VIEW;
    const size_t size = ggml_nbytes(tensor);

    CUDA_CHECK(hipSetDevice(g_system_gpu_status.main_device_id));
    if (inplace && tensor->src0->backend == GGML_BACKEND_GPU) {
        struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu * ) tensor->src0->extra;
        char * src0_ddc = (char *) src0_extra->data_device[g_system_gpu_status.main_device_id];
        size_t offset = 0;
        if (tensor->op == GGML_OP_VIEW) {
            memcpy(&offset, tensor->opt[0]->data, sizeof(size_t));
        }
        extra->data_device[g_system_gpu_status.main_device_id] = src0_ddc + offset;
    } else if (tensor->op == GGML_OP_CPY) {
        struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu * ) tensor->src1->extra;
        void * src1_ddv = src1_extra->data_device[g_system_gpu_status.main_device_id];
        extra->data_device[g_system_gpu_status.main_device_id] = src1_ddv;
    } else if (scratch) {
        GGML_ASSERT(size <= g_scratch_size);
        if (g_scratch_offset + size > g_scratch_size) {
            g_scratch_offset = 0;
        }

        char * data = (char *) g_scratch_buffer;
        if (data == nullptr) {
            CUDA_CHECK(hipMalloc(&data, g_scratch_size));
            g_scratch_buffer = data;
        }
        extra->data_device[g_system_gpu_status.main_device_id] = data + g_scratch_offset;

        g_scratch_offset += size;

        GGML_ASSERT(g_scratch_offset <= g_scratch_size);
    } else { // allocate new buffers outside of scratch
        void * data;
        CUDA_CHECK(hipMalloc(&data, size));
        CUDA_CHECK(hipMemset(data, 0, size));
        extra->data_device[g_system_gpu_status.main_device_id] = data;
    }

    tensor->extra = extra;
}

void ggml_cuda_assign_buffers(struct ggml_tensor * tensor) {
    ggml_cuda_assign_buffers_impl(tensor, true);
}

void ggml_cuda_assign_buffers_no_scratch(struct ggml_tensor * tensor) {
    ggml_cuda_assign_buffers_impl(tensor, false);
}

void ggml_cuda_set_main_device(int main_device) {
    // if (main_device >= g_system_gpu_status.num_devices) {
    //     fprintf(stderr, "warning: cannot set main_device=%d because there are only %d devices. Using device %d instead.\n",
    //             main_device, g_system_gpu_status.num_devices, g_system_gpu_status.main_device_id);
    //     return;
    // }
    // we accept setting it before initialization
    g_system_gpu_status.main_device_id = main_device;
}
void ggml_cuda_set_vram_reserved(int64_t vram_reserved_bytes) {
    for (int i = 0; i < GGML_CUDA_MAX_DEVICES; ++i)
    {
        g_system_gpu_status.device_vram_reserved[i] = vram_reserved_bytes;
    }
}

void ggml_cuda_set_scratch_size(size_t scratch_size) {
    g_scratch_size = scratch_size;
}

void ggml_cuda_free_scratch() {
    if (g_scratch_buffer == nullptr) {
        return;
    }

    CUDA_CHECK(hipFree(g_scratch_buffer));
    g_scratch_buffer = nullptr;
}

bool ggml_cuda_compute_forward(struct ggml_compute_params * params, struct ggml_tensor * tensor){
    ggml_cuda_func_t func;

    if (tensor->op == GGML_OP_NONE)
        return true;
    // user has disabled cuda or no devices found
    if (g_system_gpu_status.num_devices == 0) 
         return false;
    // allow manual skip
    if (tensor->meta.cuda_op_directive == 0) 
        return false;


    const bool any_on_device = tensor->backend == GGML_BACKEND_GPU
        || tensor->src0->backend == GGML_BACKEND_GPU || tensor->src0->backend == GGML_BACKEND_GPU_SPLIT
        || (tensor->src1 != nullptr && tensor->src1->backend == GGML_BACKEND_GPU);

    switch (tensor->op) {
        case GGML_OP_ADD:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_add;
            break;
        case GGML_OP_MUL:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_mul;
            break;
        case GGML_OP_SILU:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_silu;
            break;
        case GGML_OP_RMS_NORM:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_rms_norm;
            break;
        case GGML_OP_MUL_MAT:
            if (!any_on_device && !ggml_cuda_can_mul_mat(tensor->src0, tensor->src1, tensor)) {
                return false;
            }
            func = ggml_cuda_mul_mat;
            break;
        case GGML_OP_SCALE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_scale;
            break;
        case GGML_OP_CPY:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_cpy;
            break;
        case GGML_OP_RESHAPE:
        case GGML_OP_VIEW:
        case GGML_OP_PERMUTE:
        case GGML_OP_TRANSPOSE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_nop;
            break;
        case GGML_OP_DIAG_MASK_INF:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_diag_mask_inf;
            break;
        case GGML_OP_SOFT_MAX:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_soft_max;
            break;
        case GGML_OP_ROPE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_rope;
            break;
        default:
            return false;
    }

    if (params->ith != 0) {
        return true;
    }
    if (params->type == GGML_TASK_INIT || params->type == GGML_TASK_FINALIZE) {
        return true;
    }
    func(tensor->src0, tensor->src1, tensor);
    return true;
}